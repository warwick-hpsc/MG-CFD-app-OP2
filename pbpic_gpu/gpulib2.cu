#include "hip/hip_runtime.h"
/* CUDA utility Library */
/* written by Viktor K. Decyk, UCLA */

#include <stdlib.h>
#include <stdio.h>
/*#include <complex.h>*/
#include <cuda/std/complex>
#include "hip/hip_runtime.h"

int nblock_size = 64;
int ngrid_size = 1;
int maxgsx = 65535;
int mmcc = 0;
static int devid;

static hipError_t crc;

#define MAXSTREAMS             4
static hipStream_t streams[MAXSTREAMS] = {NULL,NULL,NULL,NULL};

__global__ void emptyKernel() {}

/*--------------------------------------------------------------------*/
void gpu_setgbsize(int nblock) {
/* set blocksize */
   nblock_size = nblock;
   return;
}

/*--------------------------------------------------------------------*/
int getmmcc() {
/* get major and minor computer capability */
   return mmcc;
}

/*--------------------------------------------------------------------*/
void gpu_fallocate(float **g_f, int nsize, int *irc) {
/* allocate global float memory on GPU, return pointer to C */
   void *gptr;
   crc = hipMalloc(&gptr,sizeof(float)*nsize);
   if (crc) {
      printf("hipMalloc float Error=%d:%s,l=%d\n",crc,
              hipGetErrorString(crc),nsize);
      *irc = 1;
   }
   *g_f = (float *)gptr;
   return;
}

/*--------------------------------------------------------------------*/
void gpu_iallocate(int **g_i, int nsize, int *irc) {
/* allocate global integer memory on GPU, return pointer to C */
   void *gptr;
   crc = hipMalloc(&gptr,sizeof(int)*nsize);
   if (crc) {
      printf("hipMalloc int Error=%d:%s,l=%d\n",crc,
              hipGetErrorString(crc),nsize);
      *irc = 1;
   }
   *g_i = (int *)gptr;
   return;
}

/*--------------------------------------------------------------------*/
void gpu_callocate(cuda::std::complex<float> **g_c, int nsize, int *irc) {
/* allocate global float memory on GPU, return pointer to C */
   void *gptr;
   crc = hipMalloc(&gptr,sizeof(cuda::std::complex<float>)*nsize);
   if (crc) {
      printf("hipMalloc cuda::std::complex<float> Error=%d:%s,l=%d\n",crc,
              hipGetErrorString(crc),nsize);
      *irc = 1;
   }
   *g_c = (cuda::std::complex<float> *)gptr;
   return;
}

/*--------------------------------------------------------------------*/
void gpu_deallocate(void *g_d, int *irc) {
/* deallocate global memory on GPU */
   crc = hipFree(g_d);
   if (crc) {
      printf("hipFree Error=%d:%s\n",crc,hipGetErrorString(crc));
      *irc = 1;
   }
   return;
}

/*--------------------------------------------------------------------*/
void hpl_fallocate(float **h_f, int nsize, int *irc) {
/* allocate page-locked float memory on host, return pointer to C */
   void *hptr = NULL;
   crc = hipHostMalloc(&hptr,sizeof(float)*nsize);
   if (crc) {
      printf("hipHostMalloc float Error=%d:%s,l=%d\n",crc,
              hipGetErrorString(crc),nsize);
      *irc = 1;
   }
   *h_f = (float *)hptr;
   return;
}

/*--------------------------------------------------------------------*/
void hpl_callocate(cuda::std::complex<float> **h_c, int nsize, int *irc) {
/* allocate page-locked float memory on host, return pointer to C */
   void *hptr = NULL;
   crc = hipHostMalloc(&hptr,sizeof(cuda::std::complex<float>)*nsize);
   if (crc) {
      printf("hipHostMalloc cuda::std::complex<float> Error=%d:%s,l=%d\n",crc,
              hipGetErrorString(crc),nsize);
      *irc = 1;
   }
   *h_c = (cuda::std::complex<float> *)hptr;
   return;
}

/*--------------------------------------------------------------------*/
void hpl_deallocate(void *h_d, int *irc) {
/* deallocate page-locked on host */
   crc = hipHostFree(h_d);
   if (crc) {
      printf("hipHostFree Error=%d:%s\n",crc,hipGetErrorString(crc));
      *irc = 1;
   }
   return;
}

/*--------------------------------------------------------------------*/
void gpu_fcopyin(float *f, float *g_f, int nsize) {
/* copy float array from host memory to global GPU memory */
   crc = hipMemcpy((void *)g_f,f,sizeof(float)*nsize,
                    hipMemcpyHostToDevice);
   if (crc) {
      printf("hipMemcpyHostToDevice float Error=%d:%s\n",crc,
              hipGetErrorString(crc));
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
void gpu_fcopyout(float *f, float *g_f, int nsize) {
/* copy float array from global GPU memory to host memory */
   crc = hipMemcpy(f,(void *)g_f,sizeof(float)*nsize,
                    hipMemcpyDeviceToHost);
   if (crc) {
      printf("hipMemcpyDeviceToHost float Error=%d:%s\n",crc,
              hipGetErrorString(crc));
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
void gpu_icopyin(int *f, int *g_f, int nsize) {
/* copy int array from host memory to global GPU memory */
   crc = hipMemcpy((void *)g_f,f,sizeof(int)*nsize,
                    hipMemcpyHostToDevice);
   if (crc) {
      printf("hipMemcpyHostToDevice int Error=%d:%s\n",crc,
              hipGetErrorString(crc));
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
void gpu_icopyout(int *f, int *g_f, int nsize) {
/* copy int array from global GPU memory to host memory */
   crc = hipMemcpy(f,(void *)g_f,sizeof(int)*nsize,
                    hipMemcpyDeviceToHost);
   if (crc) {
      printf("hipMemcpyDeviceToHost int Error=%d:%s\n",crc,
              hipGetErrorString(crc));
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
void gpu_ccopyin(cuda::std::complex<float> *f, cuda::std::complex<float> *g_f, int nsize) {
/* copy float array from host memory to global GPU memory */
   crc = hipMemcpy((void *)g_f,f,sizeof(cuda::std::complex<float>)*nsize,
                    hipMemcpyHostToDevice);
   if (crc) {
      printf("hipMemcpyHostToDevice cuda::std::complex<float> Error=%d:%s\n",crc,
              hipGetErrorString(crc));
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
void gpu_ccopyout(cuda::std::complex<float> *f, cuda::std::complex<float> *g_f, int nsize) {
/* copy cuda::std::complex<float> array from global GPU memory to host memory */
   crc = hipMemcpy(f,(void *)g_f,sizeof(cuda::std::complex<float>)*nsize,
                    hipMemcpyDeviceToHost);
   if (crc) {
      printf("hipMemcpyDeviceToHost cuda::std::complex<float> Error=%d:%s\n",crc,
              hipGetErrorString(crc));
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
void gpu_initstream(int nstream) {
/* Create Stream for requested identifier nstream       */
/* nstream should be between 1 and MAXSTREAMS inclusive */
   if ((nstream < 1) || (nstream > MAXSTREAMS)) {
      printf("gpu_initstream: nstream out of bounds = %d\n",nstream);
      exit(1);
   }
   if (streams[nstream-1] != NULL) {
      printf("gpu_initstream: nstream already used = %d\n",nstream);
      exit(1);
   }
   crc = hipStreamCreate(&streams[nstream-1]);
   if (crc) {
      printf("hipStreamCreate Error=%d:%s\n",crc,
             hipGetErrorString(crc));
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
void gpu_delstream(int nstream) {
/* Destroy Stream for requested identifier nstream      */
/* nstream should be between 1 and MAXSTREAMS inclusive */
   if ((nstream < 1) || (nstream > MAXSTREAMS)) {
      printf("gpu_delstream: nstream out of bounds = %d\n",nstream);
   }
   if (streams[nstream-1] == NULL) {
      printf("gpu_delstream: nstream not allocated = %d\n",nstream);
   }
   crc = hipStreamDestroy(streams[nstream-1]);
   if (crc) {
      printf("hipStreamDestroy Error=%d:%s\n",crc,
             hipGetErrorString(crc));
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
void gpu_waitstream(int nstream) {
/* Synchronize Stream for requested identifier nstream  */
/* nstream should be between 0 and MAXSTREAMS inclusive */
   hipStream_t stream = NULL;
   if ((nstream >= 0) || (nstream <= MAXSTREAMS)) {
      if (nstream > 0) stream = streams[nstream-1];
   }
   else {
      printf("gpu_waitstream: nstream undefined = %d\n",nstream);
      exit(1);
   }
   crc = hipStreamSynchronize(stream);
   if (crc) {
      printf("hipStreamSynchronize Error=%d:%s\n",crc,
             hipGetErrorString(crc));
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
void gpu_cascopyin(cuda::std::complex<float> *f, cuda::std::complex<float> *g_f, int noff, 
                              int nsize, int nstream) {
/* copy cuda::std::complex<float> array segment from host memory to global GPU memory */
/* asynchronous copy */
   cuda::std::complex<float> *cptr;
   hipStream_t stream = NULL;
   cptr = &g_f[noff];
   if ((nstream >= 0) || (nstream <= MAXSTREAMS)) {
      if (nstream > 0) stream = streams[nstream-1];
   }
   else {
      printf("gpu_cascopyin: nstream undefined = %d\n",nstream);
      exit(1);
   }
   crc = hipMemcpyAsync((void *)cptr,f,sizeof(cuda::std::complex<float>)*nsize,
                         hipMemcpyHostToDevice,stream);
   if (crc) {
      printf("Async hipMemcpyHostToDevice cuda::std::complex<float> Error=%d:%s\n",crc,
              hipGetErrorString(crc));
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
void gpu_cascopyout(cuda::std::complex<float> *f, cuda::std::complex<float> *g_f, int noff,
                               int nsize, int nstream) {
/* copy cuda::std::complex<float> array segment from global GPU memory to host memory */
/* asynchronous copy */
   cuda::std::complex<float> *cptr;
   hipStream_t stream = NULL;
   cptr = &g_f[noff];
   if ((nstream >= 0) || (nstream <= MAXSTREAMS)) {
      if (nstream > 0) stream = streams[nstream-1];
   }
   else {
      printf("gpu_cascopyout: nstream undefined = %d\n",nstream);
      exit(1);
   }
   crc = hipMemcpyAsync(f,(void *)cptr,sizeof(cuda::std::complex<float>)*nsize,
                         hipMemcpyDeviceToHost,stream);
   if (crc) {
      printf("Async hipMemcpyDeviceToHost cuda::std::complex<float> Error=%d:%s\n",crc,
              hipGetErrorString(crc));
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
void gpu_zfmem(float *g_f, int nsize) {
/* initialize float array in global GPU memory to zero */
   crc = hipMemset((void *)g_f,0,sizeof(float)*nsize);
   if (crc) {
      printf("hipMemset Error=%d:%s\n",crc,hipGetErrorString(crc));
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
void gpu_zcmem(cuda::std::complex<float> *g_f, int nsize) {
/* initialize cuda::std::complex<float> array in global GPU memory to zero */
   crc = hipMemset((void *)g_f,0,sizeof(cuda::std::complex<float>)*nsize);
   if (crc) {
      printf("hipMemset Error=%d:%s\n",crc,hipGetErrorString(crc));
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
void gpu_set_cache_size(int nscache) {
/* request preferred cache size, requires CUDA 3.2 or higher */
/* nscache = (0,1,2) = (no,small,big) cache size */
   hipFuncCache_t cpref;
   if ((nscache < 0) || (nscache > 2))
      return;
   if (nscache==0)
      cpref = hipFuncCachePreferNone;
   else if (nscache==1)
      cpref = hipFuncCachePreferShared;
   else if (nscache==2)
      cpref = hipFuncCachePreferL1;
   crc = hipDeviceSetCacheConfig(cpref);
/* crc = hipDeviceSetCacheConfig(cpref); */
   if (crc) {
      printf("hipDeviceSetCacheConfig error=%d:%s\n",crc,
             hipGetErrorString(crc));
   }
   return;
}

/*--------------------------------------------------------------------*/
void emptykernel() {
   int ngx, ngy;
   ngx  = nblock_size < 32768 ? nblock_size : 32768;
   ngy = (ngrid_size - 1)/ngx + 1;
   dim3 dimBlock(nblock_size,1);
   dim3 dimGrid(ngx,ngy);
   crc = hipGetLastError();
   emptyKernel<<<dimGrid,dimBlock>>>();
   hipDeviceSynchronize();
   crc = hipGetLastError();
   if (crc) {
      printf("emptyKernel error=%d:%s\n",crc,hipGetErrorString(crc));
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
void init_cu(int dev, int *irc, int proc, FILE *fp) {
/* initialize CUDA with device dev or selects best GPU available       */
/* searches throughs devices, selects the device with the most compute */
/* units, and saves the device id devid                                */
/* if dev is a valid device, it is used, otherwise the GPU with the    */
/* most multi-processors is selected                                   */
/* error code is modified only if there is an error */
   int maxcpus = 0, jm = -1;
   int j, ndevs, maxunits;
   unsigned long msize;
   double z;
   struct hipDeviceProp_t prop;
/* returns number of device */
   crc = hipGetDeviceCount(&ndevs);
   if (crc) {
      printf("hipGetDeviceCount Error=%i:%s\n",crc,
             hipGetErrorString(crc));
      *irc = 1;
      return;
   }
/* get information about devices */
   for (j = 0; j < ndevs; j++) {
      crc = hipGetDeviceProperties(&prop,j);
      if (crc) {
         printf("hipGetDeviceProperties Error=%i:%s\n",crc,
                hipGetErrorString(crc));
         prop.name[0] = 0;
      }
      maxunits = prop.multiProcessorCount;
      if (dev <= 0) {
         fprintf(fp,"j=%i:CUDA_DEVICE_NAME=%s,CUDA_MULTIPROCESSOR_COUNT=%i\n",
                j,prop.name,maxunits);
         msize = prop.totalGlobalMem;
         z = ((double) msize)/1073741824.0;
         mmcc = 10*prop.major + prop.minor;
         fprintf(fp,"    CUDA_GLOBAL_MEM_SIZE=%lu(%f GB),Capability=%d\n",
                msize,(float) z,mmcc);
         if (maxunits > maxcpus) {
            maxcpus = maxunits;
            jm = j;
         }
      }
   }
   devid = jm;
   if (dev >= 0)
      devid = dev % ndevs;
   fprintf(fp, "proc %i using device j=%i\n",proc, devid);
/* get properties for this device */
   crc = hipGetDeviceProperties(&prop,devid);
   maxgsx = prop.maxGridSize[0];
   mmcc = 10*prop.major + prop.minor;
/* set device */
   crc = hipSetDevice(devid);
   if (crc) {
      printf("hipSetDevice Error=%i:%s\n",crc,
             hipGetErrorString(crc));
      *irc = 1;
      return;
   }
/* run empty kernel */
   emptykernel();
   return;
}

void end_cu() {
/* terminate CUDA */
   crc = hipDeviceReset();
   if (crc) {
      printf("hipDeviceReset Error=%d:%s\n",crc,hipGetErrorString(crc));
   }
   return;
}
