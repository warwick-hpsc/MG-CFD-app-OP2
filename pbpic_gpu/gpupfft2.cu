#include "hip/hip_runtime.h"
/* CUDA Parallel FFT Library */
/* written by Viktor K. Decyk, UCLA */

#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>

extern int nblock_size;
extern int maxgsx;

static hipError_t crc;
static hipfftResult cfrc = HIPFFT_SUCCESS;
static hipfftHandle planrx = 0, planxr = 0, planrxn = 0, planxrn = 0;
static hipfftHandle plany = 0, planyn = 0;


__global__ void gpuppmtposes(float2 f[], float2 sm[], int nx, int kxp,
                             int kyps, int kstrt, int nvp, int kxyp,
                             int nxv, int kypd);

__global__ void gpuppmtposer(float2 g[], float2 tm[], int ny, int kyp,
                             int kxps, int kstrt, int nvp, int kxyp,
                             int nyv, int kxpd);
                             
__global__ void gpuppmtposesn(float2 fn[], float2 sm[], int nx, int kxp,
                              int kyps, int kstrt, int nvp, int ndim,
                              int kxyp, int nxv, int kypd);

__global__ void gpuppmtposern(float2 gn[], float2 tm[], int ny, int kyp,
                              int kxps, int kstrt, int nvp, int ndim,
                              int kxyp, int nyv, int kxpd);


/*--------------------------------------------------------------------*/
__global__ void gpuppsmtposes(float2 f[], float2 sm[], float ani,
                              int nx, int kxp, int kyps, int kstrt,
                              int nvp, int kxyp, int nxv, int kypd) {
/* extract data to send and normalize */
/* local data */
   int ks, j, k, n, nn, id, joff, ld;
   float2 a;
   ks = kstrt - 1;
/* for (n = 0; n < nvp; n++) { */
   n = blockIdx.y;
   if (n < nvp) {
      id = n - ks;
      if (id < 0)
         id += nvp;
/* find which node sends to itself */
      nn = 2*ks;
      if (nn >= nvp)
         nn -= nvp;
/* adjust counter */
      if (n > nn)
         n -= 1;
/* do not send local data */
      if (id != ks) {
         joff = kxp*id;
         ld = nx - joff;
         ld = 0 > ld ? 0 : ld;
         ld = kxp < ld ? kxp : ld;
/*       for (k = 0; k < kyps; k++) { */
         k = blockIdx.x;
         if (k < kyps) {
/*          for (j = 0; j < ld; j++) { */
            j = threadIdx.x;
            while (j < ld) {
               a = f[j+joff+nxv*k];
               a.x = ani*a.x;
               a.y = ani*a.y;
               sm[j+ld*k+kxyp*n] = a;
               j += blockDim.x;
            }
         }
      }
   }
   return;
}

/*--------------------------------------------------------------------*/
__global__ void gpuppsmtposesn(float2 fn[], float2 sm[], float ani,
                               int nx, int kxp, int kyps, int kstrt,
                               int nvp, int ndim, int kxyp, int nxv,
                               int kypd) {
/* extract vector data to send and normalize */
/* local data */
   int ks, i, j, k, n, nn, id, joff, ld, nnxv, nkxyp;
   float2 a;
   ks = kstrt - 1;
   nnxv = ndim*nxv;
   nkxyp = ndim*kxyp;
/* for (n = 0; n < nvp; n++) { */
   n = blockIdx.y;
   if (n < nvp) {
      id = n - ks;
      if (id < 0)
         id += nvp;
/* find which node sends to itself */
      nn = 2*ks;
      if (nn >= nvp)
         nn -= nvp;
/* adjust counter */
      if (n > nn)
         n -= 1;
/* do not send local data */
      if (id != ks) {
         joff = kxp*id;
         ld = nx - joff;
         ld = 0 > ld ? 0 : ld;
         ld = kxp < ld ? kxp : ld;
/*       for (k = 0; k < kyps; k++) { */
         k = blockIdx.x;
         if (k < kyps) {
/*          for (j = 0; j < ld; j++) { */
            j = threadIdx.x;
            while (j < ld) {
               for (i = 0; i < ndim; i++) {
                  a = fn[j+joff+nxv*i+nnxv*k];
                  a.x = ani*a.x;
                  a.y = ani*a.y;
                  sm[j+ld*(i+ndim*k)+nkxyp*n] = a;
               }
               j += blockDim.x;
            }
         }
      }
   }
   return;
}

/*--------------------------------------------------------------------*/
__global__ void gpuppsltpose(float2 f[], float2 g[], float ani, int nx,
                             int ny, int kxp, int kyp, int kstrt,
                             int nxv, int nyv) {
/* transpose local data with scaling */
/* local data */
   int mxv, j, k, ks, kxps, kyps, joff, koff, js, jj, kk;
   float2 a;
/* The size of the shared memory array is as follows: */
/* float2 s2[(mx + 1)*mx];                            */
   extern __shared__ float2 s2[];
   mxv = blockDim.x + 1;
   ks = kstrt - 1;
   joff = kxp*ks;
   koff = kyp*ks;
   kxps = nx - joff;
   kxps = 0 > kxps ? 0 : kxps;
   kxps = kxp < kxps ? kxp : kxps;
   kyps = ny - koff;
   kyps = 0 > kyps ? 0 : kyps;
   kyps = kyp < kyps ? kyp : kyps;
   js = threadIdx.x;
   ks = threadIdx.y;
   jj = blockDim.x*blockIdx.x;
   kk = blockDim.y*blockIdx.y;
   j = js + jj;
   k = ks + kk;
   if ((j < kxps) && (k < kyps)) {
      s2[js+mxv*ks] = f[j+joff+nxv*k];
   }
/* synchronize threads */
   __syncthreads();
   j = ks + jj;
   k = js + kk;
   if ((j < kxps) && (k < kyps)) {
      a = s2[ks+mxv*js];
      a.x = ani*a.x;
      a.y = ani*a.y;
      g[k+koff+nyv*j] = a;
   }
   return;
}

/*--------------------------------------------------------------------*/
__global__ void gpuppsltposen(float2 fn[], float2 gn[], float ani,
                              int nx, int ny, int kxp, int kyp,
                              int kstrt, int ndim, int nxv, int nyv) {
/* transpose local vector data with scaling */
/* local data */
   int mxv, i, j, k, ks, kxps, kyps, joff, koff, js, jj, kk;
   int nnxv, nnyv;
   float2 a;
/* The size of the shared memory array is as follows: */
/* float2 s2n[ndim*(mx + 1)*mx];                      */
   extern __shared__ float2 s2n[];
   mxv = blockDim.x + 1;
   ks = kstrt - 1;
   nnxv = ndim*nxv;
   nnyv = ndim*nyv;
   joff = kxp*ks;
   koff = kyp*ks;
   kxps = nx - joff;
   kxps = 0 > kxps ? 0 : kxps;
   kxps = kxp < kxps ? kxp : kxps;
   kyps = ny - koff;
   kyps = 0 > kyps ? 0 : kyps;
   kyps = kyp < kyps ? kyp : kyps;
   js = threadIdx.x;
   ks = threadIdx.y;
   jj = blockDim.x*blockIdx.x;
   kk = blockDim.y*blockIdx.y;
   j = js + jj;
   k = ks + kk;
   if ((j < kxps) && (k < kyps)) {
      for (i = 0; i < ndim; i++) {
         s2n[js+mxv*(i+ndim*ks)] = fn[j+joff+nxv*i+nnxv*k];
      }
   }
/* synchronize threads */
   __syncthreads();
   j = ks + jj;
   k = js + kk;
   if ((j < kxps) && (k < kyps)) {
      for (i = 0; i < ndim; i++) {
         a = s2n[ks+mxv*(i+ndim*js)];
         a.x = ani*a.x;
         a.y = ani*a.y;
         gn[k+koff+nyv*i+nnyv*j] = a;
      }
   }
   return;
}

/*--------------------------------------------------------------------*/	
extern "C" void gpupfft2rrcuinit(int nx, int kypp, int ndim) {
   if (kypp <= 0)
      return;
   cfrc = hipfftPlan1d(&planrx,nx,HIPFFT_R2C,kypp);
   if (cfrc) {
      printf("hipfftPlan1d planrx error=%d\n",cfrc);
      exit(1);
   }
   cfrc = hipfftPlan1d(&planxr,nx,HIPFFT_C2R,kypp);
   if (cfrc) {
      printf("hipfftPlan1d planxr error=%d\n",cfrc);
      exit(1);
   }
   cfrc = hipfftPlan1d(&planrxn,nx,HIPFFT_R2C,ndim*kypp);
   if (cfrc) {
      printf("hipfftPlan1d planrxn error=%d\n",cfrc);
      exit(1);
   }
   cfrc = hipfftPlan1d(&planxrn,nx,HIPFFT_C2R,ndim*kypp);
   if (cfrc) {
      printf("hipfftPlan1d planxrn error=%d\n",cfrc);
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/	
extern "C" void gpupfft2cuinit(int kxpp, int ny, int ndim) {
   if (kxpp <= 0)
      return;
   cfrc = hipfftPlan1d(&plany,ny,HIPFFT_C2C,kxpp);
   if (cfrc) {
      printf("hipfftPlan1d plany error=%d\n",cfrc);
      exit(1);
   }
   cfrc = hipfftPlan1d(&planyn,ny,HIPFFT_C2C,ndim*kxpp);
   if (cfrc) {
      printf("hipfftPlan1d planyn error=%d\n",cfrc);
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpupfft2rrcudel() {
   if (planrx != 0)
      cfrc = hipfftDestroy(planrx);
   if (cfrc) {
      printf("hipfftPlan1d planrx error=%d\n",cfrc);
      exit(1);
   }
   if (planxr != 0)
      cfrc = hipfftDestroy(planxr);
   if (cfrc) {
      printf("hipfftPlan1d planxr error=%d\n",cfrc);
      exit(1);
   }
   if (planrxn != 0)
      cfrc = hipfftDestroy(planrxn);
   if (cfrc) {
      printf("hipfftPlan1d planrxn error=%d\n",cfrc);
      exit(1);
   }
   if (planxr != 0)
      cfrc = hipfftDestroy(planxrn);
   if (cfrc) {
      printf("hipfftPlan1d planxrn error=%d\n",cfrc);
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpupfft2cudel() {
   if (plany != 0)
      cfrc = hipfftDestroy(plany);
   if (cfrc) {
      printf("hipfftPlan1d plany error=%d\n",cfrc);
      exit(1);
   }
   if (planyn != 0)
      cfrc = hipfftDestroy(planyn);
   if (cfrc) {
      printf("hipfftPlan1d planyn error=%d\n",cfrc);
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpupfft2rrcux(float2 *f, float2 *bsm, int isign, 
                              int indx, int indy, int kstrt, int nvp,
                              int kxp1, int kyp, int nxh1d, int kypd) {
/* wrapper function for real to complex fft in x,                */
/* without packed data                                           */
/* uses 1D real to complex and complex to complex NVIDIA FFTs    */
/* nxh1d must be = nx/2+1                                        */
/* local data */
   int nx, nxh1, ny, ks, kypp, kxyp, ns;
   int mx = 16;
   float ani;
   dim3 dimBlock(nblock_size);
   dim3 dimBlockt(mx,mx);
/* calculate range of indices */
   nx = 1L<<indx;
   nxh1 = nx/2 + 1;
   ny = 1L<<indy;
   ks = kstrt - 1;
   kypp = ny - kyp*ks;
   kypp = 0 > kypp ? 0 : kypp;
   kypp = kyp < kypp ? kyp : kypp;
   if (kypp <= 0)
      return;
   kxyp = kxp1*kyp;
   dim3 dimGrids(kypp,nvp);
   dim3 dimGridty((kyp-1)/mx+1,(kxp1-1)/mx+1,nvp);
   ns = (mx+1)*mx*sizeof(float2);
/* inverse fourier transform */
   if (isign < 0) {
/* perform x fft */
      cfrc = hipfftExecR2C(planrx,(hipfftReal *)f,(hipfftComplex *)f);
/*    hipDeviceSynchronize(); */
      if (cfrc) {
         printf("hipfftExecR2C(-1) planrx error=%d\n",cfrc);
         exit(1);
      }
/* extract data to send and normalize */
      ani = 1.0f/(((float) nx)*((float) ny));
      crc = hipGetLastError();
      gpuppsmtposes<<<dimGrids,dimBlock>>>(f,bsm,ani,nxh1,kxp1,kypp,
                                           kstrt,nvp,kxyp,nxh1d,kypd);
      hipDeviceSynchronize();
      crc = hipGetLastError();
      if (crc) {
         printf("gpuppsmtposes error=%d:%s\n",crc,
                hipGetErrorString(crc));
         exit(1);
      }
   }
/* forward fourier transform */
   else if (isign > 0) {
/* transpose data received */
      crc = hipGetLastError();
      gpuppmtposer<<<dimGridty,dimBlockt,ns>>>(f,bsm,nxh1,kxp1,kypp,
                                               kstrt,nvp,kxyp,nxh1d,
                                               kypd);
/*    hipDeviceSynchronize(); */
      crc = hipGetLastError();
      if (crc) {
         printf("gpuppmtposer error=%d:%s\n",crc,
                hipGetErrorString(crc));
         exit(1);
      }
/* perform x fft */
      cfrc = hipfftExecC2R(planxr,(hipfftComplex *)f,(hipfftReal *)f);
      hipDeviceSynchronize();
      if (cfrc) {
         printf("hipfftExecC2R(1) planxr error=%d\n",cfrc);
         exit(1);
      }
   }
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpupfft2rrcuy(float2 *g, float2 *brm, int isign,
                              int indx, int indy, int kstrt, int nvp, 
                              int kxp1, int kyp, int nyd) {
/* wrapper function for real to complex fft in y,                */
/* without packed data                                           */
/* uses 1D real to complex and complex to complex NVIDIA FFTs    */
/* local data */
   int nx, nxh1, ny, ks, kxpp, kxyp, ns;
   int mx = 16;
   dim3 dimBlock(nblock_size);
   dim3 dimBlockt(mx,mx);
/* calculate range of indices */
   nx = 1L<<indx;
   nxh1 = nx/2 + 1;
   ny = 1L<<indy;
   ks = kstrt - 1;
   kxpp = nxh1 - kxp1*ks;
   kxpp = 0 > kxpp ? 0 : kxpp;
   kxpp = kxp1 < kxpp ? kxp1 : kxpp;
   if (kxpp <= 0)
      return;
   kxyp = kxp1*kyp;
   dim3 dimGrids(kxpp,nvp);
   dim3 dimGridtx((kxp1-1)/mx+1,(kyp-1)/mx+1,nvp);
   ns = (mx+1)*mx*sizeof(float2);
/* inverse fourier transform */
   if (isign < 0) {
/* transpose data received */
      crc = hipGetLastError();
      gpuppmtposer<<<dimGridtx,dimBlockt,ns>>>(g,brm,ny,kyp,kxpp,kstrt,
                                               nvp,kxyp,nyd,kxp1);
/*    hipDeviceSynchronize(); */
      crc = hipGetLastError();
      if (crc) {
         printf("gpuppmtposer error=%d:%s\n",crc,
                hipGetErrorString(crc));
         exit(1);
      }
/* perform y fft */
      cfrc = hipfftExecC2C(plany,(hipfftComplex *)g,(hipfftComplex *)g,
                          HIPFFT_FORWARD);
      hipDeviceSynchronize();
      if (cfrc) {
         printf("hipfftExecC2C(-1) plany error=%d\n",cfrc);
         exit(1);
      }
   }
/* forward fourier transform */
   else if (isign > 0) {
/* perform y fft */
      cfrc = hipfftExecC2C(plany,(hipfftComplex *)g,(hipfftComplex *)g,
                          HIPFFT_BACKWARD);
/*    hipDeviceSynchronize(); */
      if (cfrc) {
         printf("hipfftExecC2C(1) plany error=%d\n",cfrc);
         exit(1);
      }
/* extract data to send */
      crc = hipGetLastError();
      gpuppmtposes<<<dimGrids,dimBlock>>>(g,brm,ny,kyp,kxpp,kstrt,
                                          nvp,kxyp,nyd,kxp1);
      hipDeviceSynchronize();
      crc = hipGetLastError();
      if (crc) {
         printf("gpuppmtposes error=%d:%s\n",crc,
                hipGetErrorString(crc));
         exit(1);
      }
   }
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpupfft2rrcuxn(float2 *fn, float2 *bsm, int isign, 
                               int indx, int indy, int ndim, int kstrt,
                               int nvp, int kxp1, int kyp, int nxh1d,
                               int kypd) {
/* wrapper function for real to complex fft in x,                */
/* without packed data                                           */
/* uses 1D real to complex and complex to complex NVIDIA FFTs    */
/* ndim = vector dimension                                       */
/* nxh1d must be = nx/2+1                                        */
/* local data */
   int nx, nxh1, ny, ks, kypp, kxyp, ns;
   int mx = 16;
   float ani;
   dim3 dimBlock(nblock_size);
   dim3 dimBlockt(mx,mx);
/* calculate range of indices */
   nx = 1L<<indx;
   nxh1 = nx/2 + 1;
   ny = 1L<<indy;
   ks = kstrt - 1;
   kypp = ny - kyp*ks;
   kypp = 0 > kypp ? 0 : kypp;
   kypp = kyp < kypp ? kyp : kypp;
   if (kypp <= 0)
      return;
   kxyp = kxp1*kyp;
   dim3 dimGrids(kypp,nvp);
   dim3 dimGridty((kyp-1)/mx+1,(kxp1-1)/mx+1,nvp);
   ns = ndim*(mx+1)*mx*sizeof(float2);
/* inverse fourier transform */
   if (isign < 0) {
/* perform x fft */
      cfrc = hipfftExecR2C(planrxn,(hipfftReal *)fn,(hipfftComplex *)fn);
/*    hipDeviceSynchronize(); */
      if (cfrc) {
         printf("hipfftExecR2C(-1) planrxn error=%d\n",cfrc);
         exit(1);
      }
/* extract data to send and normalize */
      ani = 1.0f/(((float) nx)*((float) ny));
      crc = hipGetLastError();
      gpuppsmtposesn<<<dimGrids,dimBlock>>>(fn,bsm,ani,nxh1,kxp1,kypp,
                                            kstrt,nvp,ndim,kxyp,nxh1d,
                                            kypd);
      hipDeviceSynchronize();
      crc = hipGetLastError();
      if (crc) {
         printf("gpuppsmtposesn error=%d:%s\n",crc,
                hipGetErrorString(crc));
         exit(1);
      }
   }
/* forward fourier transform */
   else if (isign > 0) {
/* transpose data received */
      crc = hipGetLastError();
      gpuppmtposern<<<dimGridty,dimBlockt,ns>>>(fn,bsm,nxh1,kxp1,kypp,
                                                kstrt,nvp,ndim,kxyp,
                                                nxh1d,kypd);
/*    hipDeviceSynchronize(); */
      crc = hipGetLastError();
      if (crc) {
         printf("gpuppmtposern error=%d:%s\n",crc,
                hipGetErrorString(crc));
         exit(1);
      }
/* perform x fft */
      cfrc = hipfftExecC2R(planxrn,(hipfftComplex *)fn,(hipfftReal *)fn);
      hipDeviceSynchronize();
      if (cfrc) {
         printf("hipfftExecC2R(1) planxrn error=%d\n",cfrc);
         exit(1);
      }
   }
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpupfft2rrcuyn(float2 *gn, float2 *brm, int isign,
                               int indx, int indy, int ndim, int kstrt,
                               int nvp, int kxp1, int kyp, int nyd) {
/* wrapper function for real to complex fft in y,                */
/* without packed data                                           */
/* uses 1D real to complex and complex to complex NVIDIA FFTs    */
/* ndim = vector dimension                                       */
/* local data */
   int nx, nxh1, ny, ks, kxpp, kxyp, ns;
   int mx = 16;
   dim3 dimBlock(nblock_size);
   dim3 dimBlockt(mx,mx);
/* calculate range of indices */
   nx = 1L<<indx;
   nxh1 = nx/2 + 1;
   ny = 1L<<indy;
   ks = kstrt - 1;
   kxpp = nxh1 - kxp1*ks;
   kxpp = 0 > kxpp ? 0 : kxpp;
   kxpp = kxp1 < kxpp ? kxp1 : kxpp;
   if (kxpp <= 0)
      return;
   kxyp = kxp1*kyp;
   dim3 dimGrids(kxpp,nvp);
   dim3 dimGridtx((kxp1-1)/mx+1,(kyp-1)/mx+1,nvp);
   ns = ndim*(mx+1)*mx*sizeof(float2);
/* inverse fourier transform */
   if (isign < 0) {
/* transpose data received */
      crc = hipGetLastError();
      gpuppmtposern<<<dimGridtx,dimBlockt,ns>>>(gn,brm,ny,kyp,kxpp,kstrt,
                                                nvp,ndim,kxyp,nyd,kxp1);
/*    hipDeviceSynchronize(); */
      crc = hipGetLastError();
      if (crc) {
         printf("gpuppmtposern error=%d:%s\n",crc,
                hipGetErrorString(crc));
         exit(1);
      }
/* perform y fft */
      cfrc = hipfftExecC2C(planyn,(hipfftComplex *)gn,(hipfftComplex *)gn,
                          HIPFFT_FORWARD);
      hipDeviceSynchronize();
      if (cfrc) {
         printf("hipfftExecC2C(-1) planyn error=%d\n",cfrc);
         exit(1);
      }
   }
/* forward fourier transform */
   else if (isign > 0) {
/* perform y fft */
      cfrc = hipfftExecC2C(planyn,(hipfftComplex *)gn,(hipfftComplex *)gn,
                          HIPFFT_BACKWARD);
/*    hipDeviceSynchronize(); */
      if (cfrc) {
         printf("hipfftExecC2C(1) planyn error=%d\n",cfrc);
         exit(1);
      }
/* extract data to send */
      crc = hipGetLastError();
      gpuppmtposesn<<<dimGrids,dimBlock>>>(gn,brm,ny,kyp,kxpp,kstrt,
                                           nvp,ndim,kxyp,nyd,kxp1);
      hipDeviceSynchronize();
      crc = hipGetLastError();
      if (crc) {
         printf("gpuppmtposesn error=%d:%s\n",crc,
                hipGetErrorString(crc));
         exit(1);
      }
   }
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void cgpuppsltpose(float2 *f, float2 *g, float ani, int nx,
                              int ny, int kxp, int kyp, int kstrt,
                              int nxv, int nyv) {
/* local complex transpose with scaling */
/* input = f, output = g                */
/* local data */
   int ns;
   static int mx = 16;
   dim3 dimBlockt(mx,mx);
/* calculate range of indices */
   dim3 dimGridtx((kxp-1)/mx+1,(kyp-1)/mx+1);
   ns = (mx+1)*mx*sizeof(float2);
/* local transpose f to g */
   crc = hipGetLastError();
   gpuppsltpose<<<dimGridtx,dimBlockt,ns>>>(f,g,ani,nx,ny,kxp,kyp,kstrt,
                                            nxv,nyv);
   hipDeviceSynchronize();
   crc = hipGetLastError();
   if (crc) {
      printf("gpuppsltpose error=%d:%s\n",crc,hipGetErrorString(crc));
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void cgpuppsltposen(float2 *fn, float2 *gn, float ani,
                               int nx, int ny, int kxp, int kyp,
                               int kstrt, int ndim, int nxv, int nyv) {
/* local complex vector transpose with scaling */
/* input = fn, output = gn                     */
/* local data */
   int ns;
   static int mx = 16;
   dim3 dimBlockt(mx,mx);
/* calculate range of indices */
   dim3 dimGridtx((kxp-1)/mx+1,(kyp-1)/mx+1);
   ns = ndim*(mx+1)*mx*sizeof(float2);
/* local transpose f to g */
   crc = hipGetLastError();
   gpuppsltposen<<<dimGridtx,dimBlockt,ns>>>(fn,gn,ani,nx,ny,kxp,kyp,
                                             kstrt,ndim,nxv,nyv);
   hipDeviceSynchronize();
   crc = hipGetLastError();
   if (crc) {
      printf("gpuppsltposen error=%d:%s\n",crc,hipGetErrorString(crc));
      exit(1);
   }
   return;
}

/* Interfaces to Fortran */

/*--------------------------------------------------------------------*/	
extern "C" void gpupfft2rrcuinit_(int *nx, int *kypp, int *ndim) {
   gpupfft2rrcuinit(*nx,*kypp,*ndim);
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpupfft2cuinit_(int *nx, int *ny, int *ndim) {
   gpupfft2cuinit(*nx,*ny,*ndim);
   return;
}


/*--------------------------------------------------------------------*/
extern "C" void gpupfft2rrcudel_() {
   gpupfft2rrcudel();
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpupfft2cudel_() {
   gpupfft2cudel();
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpupfft2rrcux_(unsigned long *gp_f, 
                               unsigned long *gp_bsm, int *isign,
                               int *indx, int *indy, int *kstrt,
                               int *nvp, int *kxp1, int *kyp,
                               int *nxh1d, int *kypd) {
   float2 *f, *bsm;
   f = (float2 *)*gp_f;
   bsm = (float2 *)*gp_bsm;
   gpupfft2rrcux(f,bsm,*isign,*indx,*indy,*kstrt,*nvp,*kxp1,*kyp,*nxh1d,
                *kypd);
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpupfft2rrcuy_(unsigned long *gp_g, 
                               unsigned long *gp_brm, int *isign,
                               int *indx, int *indy, int *kstrt,
                               int *nvp, int *kxp1, int *kyp,
                               int *nyd) {
   float2 *g, *brm;
   g = (float2 *)*gp_g;
   brm = (float2 *)*gp_brm;
   gpupfft2rrcuy(g,brm,*isign,*indx,*indy,*kstrt,*nvp, *kxp1,*kyp,*nyd);
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpupfft2rrcuxn_(unsigned long *gp_fn, 
                                unsigned long *gp_bsm, int *isign,

                               int *indx, int *indy, int *ndim, int *kstrt,
                               int *nvp, int *kxp1, int *kyp, int *nxh1d,
                               int *kypd) {
   float2 *fn, *bsm;
   fn = (float2 *)*gp_fn;
   bsm = (float2 *)*gp_bsm;
   gpupfft2rrcuxn(fn,bsm,*isign,*indx,*indy,*ndim,*kstrt,*nvp,*kxp1,
                  *kyp,*nxh1d,*kypd);
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpupfft2rrcuyn_(unsigned long *gp_gn, 
                                unsigned long *gp_brm, int *isign,
                                int *indx, int *indy, int *ndim,
                                int *kstrt, int *nvp, int *kxp1,
                                int *kyp, int *nyd) {
   float2 *gn, *brm;
   gn = (float2 *)*gp_gn;
   brm = (float2 *)*gp_brm;
   gpupfft2rrcuyn(gn,brm,*isign,*indx,*indy,*ndim,*kstrt,*nvp,*kxp1,
                  *kyp,*nyd);
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void cgpuppsltpose_(unsigned long *gp_f, unsigned long *gp_g,
                               float *ani, int *nx, int *ny, int *kxp,
                               int *kyp, int *kstrt, int *nxv,
                               int *nyv) {
   float2 *f, *g;
   f = (float2 *)*gp_f;
   g = (float2 *)*gp_g;
   cgpuppsltpose(f,g,*ani,*nx,*ny,*kxp,*kyp,*kstrt,*nxv,*nyv);
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void cgpuppsltposen_(unsigned long *gp_fn,
                                unsigned long *gp_gn, float *ani,
                                int *nx, int *ny, int *kxp, int *kyp,
                                int *kstrt, int *ndim, int *nxv,
                                int *nyv) {
   float2 *fn, *gn;
   fn = (float2 *)*gp_fn;
   gn = (float2 *)*gp_gn;
   cgpuppsltposen(fn,gn,*ani,*nx,*ny,*kxp,*kyp,*kstrt,*ndim,*nxv,*nyv);
   return;
}